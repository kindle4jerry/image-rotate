#include "hip/hip_runtime.h"
/*
 * \file cuda_edge.cu
 */
#include "cuda_edge.hpp"
#include "timer.hpp"
__device__
inline unsigned un_ceil(unsigned const a, unsigned const b) {return (a+b-1)/b; }

// Kernel that calcsize_tates a B&W image from an RGB image
// ressize_tting image has a double type for each pixel position
__global__
void bw_kernel(double *bwMat, unsigned char *gpuMat, unsigned nHpix)
{
	unsigned nThdsPerBlk = blockDim.x;
	unsigned bIdx = blockIdx.x;
	unsigned tIdx = threadIdx.x;
	unsigned gtIdx = nThdsPerBlk * bIdx + tIdx;
	double R, G, B;

	unsigned BlkPerRow = un_ceil(nHpix, nThdsPerBlk);
	unsigned RowBytes = (nHpix * 3 + 3) & (~3);
	unsigned rowBgn = bIdx / BlkPerRow;
	unsigned colBgn = gtIdx - rowBgn*BlkPerRow*nThdsPerBlk;
	if (colBgn >= nHpix) return;			// col out of range

	unsigned srcIdx = rowBgn * RowBytes + 3 * colBgn;
	unsigned pixIdx = rowBgn * nHpix + colBgn;

	B = (double)gpuMat[srcIdx];
	G = (double)gpuMat[srcIdx + 1];
	R = (double)gpuMat[srcIdx + 2];
	bwMat[pixIdx] = (R+G+B)/3.0;
}

__device__
double Gauss[5][5] = {	{ 2, 4,  5,  4,  2 },
						{ 4, 9,  12, 9,  4 },
						{ 5, 12, 15, 12, 5 },
						{ 4, 9,  12, 9,  4 },
						{ 2, 4,  5,  4,  2 } };
// Kernel that calcsize_tates a Gauss image from the B&W image
// ressize_tting image has a double type for each pixel position
__global__
void gauss_kernel(double *gaussMat, double *bwMat, unsigned nHpix, unsigned nVpix)
{
	unsigned nThdsPerBlk = blockDim.x;
	unsigned bIdx = blockIdx.x;
	unsigned tIdx = threadIdx.x;
	unsigned gtIdx = nThdsPerBlk * bIdx + tIdx;

	unsigned BlkPerRow = un_ceil(nHpix, nThdsPerBlk);
	int rowBgn = bIdx / BlkPerRow;
	int colBgn = gtIdx - rowBgn*BlkPerRow*nThdsPerBlk;
	if (colBgn >= nHpix) return;			// col out of range

	unsigned pixIdx = rowBgn * nHpix + colBgn;
	if ((rowBgn<2) || (rowBgn>nVpix - 3) || (colBgn<2) || (colBgn>nHpix - 3)){
		gaussMat[pixIdx] = 0.0;
		return;
	}else{
		double G = 0.0;
		for (int i = -2; i <= 2; i++){
			for (int j = -2; j <= 2; j++){
				int row = rowBgn + i;
				int col = colBgn + j;
				int indx = row*nHpix + col;
				G += (bwMat[indx] * Gauss[i + 2][j + 2]);
			}
		}
		gaussMat[pixIdx] = G / 159.00;
	}
}


__device__
double Gx[3][3] = { { -1, 0, 1 },
					{ -2, 0, 2 },
					{ -1, 0, 1 } };
__device__
double Gy[3][3] = { { -1, -2, -1 },
					{ 0, 0, 0 },
					{ 1, 2, 1 } };
// Kernel that calcsize_tates Gradient, Theta from the Gauss image
// ressize_tting image has a double type for each pixel position
__global__
void sobel_kernel(double *gradMat, double *thetaMat, double *gaussMat, unsigned nHpix, unsigned nVpix)
{
	unsigned nThdsPerBlk = blockDim.x;
	unsigned bIdx = blockIdx.x;
	unsigned tIdx = threadIdx.x;
	unsigned gtIdx = nThdsPerBlk * bIdx + tIdx;

	unsigned BlkPerRow = un_ceil(nHpix, nThdsPerBlk);
	int rowBgn = bIdx / BlkPerRow;
	int colBgn = gtIdx - rowBgn*BlkPerRow*nThdsPerBlk;
	if (colBgn >= nHpix) return;			// col out of range

	unsigned pixIdx = rowBgn * nHpix + colBgn;
	if ((rowBgn<1) || (rowBgn>nVpix - 2) || (colBgn<1) || (colBgn>nHpix - 2)){
		gradMat[pixIdx] = 0.0;
		thetaMat[pixIdx] = 0.0;
		return;
	}else{
		double GX = 0.0;  double GY = 0.0;
		for (int i = -1; i <= 1; i++){
			for (int j = -1; j <= 1; j++){
				int row = rowBgn + i;
				int col = colBgn + j;
				int indx = row*nHpix + col;
				GX += (gaussMat[indx] * Gx[i + 1][j + 1]);
				GY += (gaussMat[indx] * Gy[i + 1][j + 1]);
			}
		}
		gradMat[pixIdx] = sqrt(GX*GX + GY*GY);
		thetaMat[pixIdx] = atan(GX / GY)*180.0 / PI;
	}
}


// Kernel that calcsize_tates the threshold image from Gradient, Theta
// ressize_tting image has an RGB for each pixel, same RGB for each pixel
__global__
void prune_kernel(unsigned char *edgeImg, double *gradMat, double *thetaMat, unsigned nHpix, unsigned nVpix, unsigned ThreshLo, unsigned ThreshHi)
{
	unsigned nThdsPerBlk = blockDim.x;
	unsigned bIdx = blockIdx.x;
	unsigned tIdx = threadIdx.x;
	unsigned gtIdx = nThdsPerBlk * bIdx + tIdx;

	unsigned BlkPerRow = un_ceil(nHpix, nThdsPerBlk);
	unsigned RowBytes = (nHpix * 3 + 3) & (~3);
	int rowBgn = bIdx / BlkPerRow;
	int colBgn = gtIdx - rowBgn*BlkPerRow*nThdsPerBlk;
	if (colBgn >= nHpix) return;			// col out of range

	unsigned imgIdx = rowBgn * RowBytes + 3 * colBgn;
	unsigned pixIdx = rowBgn * nHpix + colBgn;
	if ((rowBgn<1) || (rowBgn>nVpix - 2) || (colBgn<1) || (colBgn>nHpix - 2)){
		edgeImg[imgIdx] = EFALSE;
		edgeImg[imgIdx + 1] = EFALSE;
		edgeImg[imgIdx + 2] = EFALSE;
		return;
	}else{
	unsigned char ePix; // edge or not
	double L, H, G, T;
		L = (double)ThreshLo;		H = (double)ThreshHi;
		G = gradMat[pixIdx];
		ePix = EFALSE;
		if (G <= L){						// no edge
			ePix = EFALSE;
		}else if (G >= H){					// edge
			ePix = ETRUE;
		}else{
			T = thetaMat[pixIdx];
			if ((T<-67.5) || (T>67.5)){
				// Look at left and right: [row][col-1]  and  [row][col+1]
				ePix = ((gradMat[pixIdx - 1]>H) || (gradMat[pixIdx + 1]>H)) ? ETRUE : EFALSE;
			}
			else if ((T >= -22.5) && (T <= 22.5)){
				// Look at top and bottom: [row-1][col]  and  [row+1][col]
				ePix = ((gradMat[pixIdx - nHpix]>H) || (gradMat[pixIdx + nHpix]>H)) ? ETRUE : EFALSE;
			}
			else if ((T>22.5) && (T <= 67.5)){
				// Look at upper right, lower left: [row-1][col+1]  and  [row+1][col-1]
				ePix = ((gradMat[pixIdx - nHpix + 1]>H) || (gradMat[pixIdx + nHpix - 1]>H)) ? ETRUE : EFALSE;
			}
			else if ((T >= -67.5) && (T<-22.5)){
				// Look at upper left, lower right: [row-1][col-1]  and  [row+1][col+1]
				ePix = ((gradMat[pixIdx - nHpix - 1]>H) || (gradMat[pixIdx + nHpix + 1]>H)) ? ETRUE : EFALSE;
			}
		}
		edgeImg[imgIdx] = ePix;
		edgeImg[imgIdx + 1] = ePix;
		edgeImg[imgIdx + 2] = ePix;
	}
}

int launch_edge_kernel(CImageBMP &image, unsigned const ThreshLo, unsigned const ThreshHi, unsigned const nThdsPerBlk) 
{
    // Choose which GPU to run on, change this on a msize_tti-GPU system.
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		cout <<"\nNo CUDA Device is available\n\n";
		return 1; 
	}
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout <<"\ncudaSetDevice failed!  Do you have a CUDA-capable GPU installed?\n\n";
		return 1;
	}
	
    hipDeviceProp_t GPUprop;
	hipGetDeviceProperties(&GPUprop, 0);
	size_t SupportedKBlocks = (unsigned) GPUprop.maxGridSize[0] * (unsigned) GPUprop.maxGridSize[1] * (unsigned )GPUprop.maxGridSize[2]/1024;
	size_t SupportedMBlocks = SupportedKBlocks / 1024;
    size_t outB =  (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks;
    char outC = (SupportedMBlocks >= 5) ? 'M' : 'K';
    string SupportedBlocks(to_string(outB)); 
    SupportedBlocks = SupportedBlocks +" " + outC; 
	size_t maxThdsPerBlk = (unsigned)GPUprop.maxThreadsPerBlock;

	hipEvent_t time1, time2, time2BW, time2Gauss, time2Sobel, time3, time4;
	hipEventCreate(&time1);		hipEventCreate(&time2);	
	hipEventCreate(&time2BW);		hipEventCreate(&time2Gauss);	hipEventCreate(&time2Sobel);	
	hipEventCreate(&time3);		hipEventCreate(&time4);

	hipEventRecord(time1, 0);		// Time stamp at the start of the GPU transfer
    // Allocate GPU buffer for the input and output images and the intermediate result
    size_t const IMAGEPIX  = (image.nHpix*image.nVpix);
    size_t const IMAGESIZE = IMAGEPIX* sizeof(CPixel); //image.nHpix * image.nVpix * sizeof(CPixel); 
    size_t GPUtotalBufferSize = 4 * sizeof(double)*IMAGEPIX + 2 * sizeof(unsigned char)*IMAGESIZE;
	
	Timer t;
	void *ptrGPU;			// Pointer to the bulk-allocated GPU memory
    cudaStatus = hipMalloc((void**)&ptrGPU, GPUtotalBufferSize);
    if (cudaStatus != hipSuccess) {
		cout <<"\ncudaMalloc failed! Can't allocate GPU memory\n\n";
		return 1; 
	}
	t.printDiff("Malloc time: ");
	unsigned char *imgGPU, *ptrImgGPU;	// Where images are stored in GPU
	imgGPU			= (unsigned char *)ptrGPU;
	ptrImgGPU	= imgGPU + IMAGESIZE;

    double  *GPUBWImg, *GPUGaussImg, *gradGPU, *thetaGPU;	
	GPUBWImg	= (double *)(ptrImgGPU + IMAGESIZE);
	GPUGaussImg	= GPUBWImg + IMAGEPIX;
	gradGPU		= GPUGaussImg + IMAGEPIX;
	thetaGPU	= gradGPU + IMAGEPIX;

	t.printDiff("Memcpystart time: ");
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(imgGPU, image.pixMat.data(), IMAGESIZE, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cout <<"\ncudaMemcpy  CPU to GPU  failed!\n\n";
		return 1; 
	}
	hipEventRecord(time2, 0);		// Time stamp after the CPU --> GPU tfr is done
	t.printDiff("Memcpy1 time: ");
	
	unsigned BlkPerRow = host_ceil(image.nHpix, nThdsPerBlk);
	unsigned nBlks = image.nVpix*BlkPerRow;
	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	bw_kernel <<< nBlks, nThdsPerBlk >>> (GPUBWImg, imgGPU, image.nHpix);
	if ((cudaStatus = hipDeviceSynchronize()) != hipSuccess) return 1; //goto KERNELERROR;
	hipEventRecord(time2BW, 0);		// Time stamp after BW image calcsize_tation
	unsigned GPUDataTfrBW, GPUDataTfrGauss, GPUDataTfrSobel, GPUDataTfrThresh,GPUDataTfrKernel, GPUDataTfrTotal;
	GPUDataTfrBW = sizeof(double)*IMAGEPIX + sizeof(unsigned char)*IMAGESIZE;

	gauss_kernel <<< nBlks, nThdsPerBlk >>> (GPUGaussImg, GPUBWImg, image.nHpix, image.nVpix);
	if ((cudaStatus = hipDeviceSynchronize()) != hipSuccess) return 1; //goto KERNELERROR; 
	hipEventRecord(time2Gauss, 0);		// Time stamp after Gauss image calcsize_tation
	GPUDataTfrGauss = 2*sizeof(double)*IMAGEPIX;

	sobel_kernel <<< nBlks, nThdsPerBlk >>> (gradGPU, thetaGPU, GPUGaussImg, image.nHpix, image.nVpix);
	if ((cudaStatus = hipDeviceSynchronize()) != hipSuccess) return 1; //goto KERNELERROR; 
	hipEventRecord(time2Sobel, 0);		// Time stamp after Gradient, Theta computation
	GPUDataTfrSobel = 3 * sizeof(double)*IMAGEPIX;

	prune_kernel <<< nBlks, nThdsPerBlk >>> (ptrImgGPU, gradGPU, thetaGPU, image.nHpix, image.nVpix, ThreshLo, ThreshHi);
	if ((cudaStatus = hipDeviceSynchronize()) != hipSuccess) return 1; //goto KERNELERROR;
	GPUDataTfrThresh = sizeof(double)*IMAGEPIX + sizeof(unsigned char)*IMAGESIZE;
	GPUDataTfrKernel = GPUDataTfrBW + GPUDataTfrGauss + GPUDataTfrSobel + GPUDataTfrThresh;
	GPUDataTfrTotal = GPUDataTfrKernel + 2 * IMAGESIZE;
	hipEventRecord(time3, 0);

	t.printDiff("Memcpystart time: ");
	// Copy output (ressize_tts) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(image.pixMat.data(), ptrImgGPU, IMAGESIZE, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cout <<"\ncudaMemcpy GPU to CPU  failed! \n\n";
		return 1; //goto EXITCUDAERROR;
	}
	hipEventRecord(time4, 0);
	t.printDiff("Memcpy2 time: ");

	hipEventSynchronize(time1);	hipEventSynchronize(time2);
	hipEventSynchronize(time2BW);	hipEventSynchronize(time2Gauss);	hipEventSynchronize(time2Sobel);
	hipEventSynchronize(time3);	hipEventSynchronize(time4);

    float totalTime, tfrCPUtoGPU, tfrGPUtoCPU;
	float kernelExecTimeBW, kernelExecTimeGauss, kernelExecTimeSobel, kernelExecTimeThreshold;
	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecTimeBW, time2, time2BW);
	hipEventElapsedTime(&kernelExecTimeGauss, time2BW, time2Gauss);
	hipEventElapsedTime(&kernelExecTimeSobel, time2Gauss, time2Sobel);
	hipEventElapsedTime(&kernelExecTimeThreshold, time2Sobel, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);
	float totalKernelTime = kernelExecTimeBW + kernelExecTimeGauss + kernelExecTimeSobel + kernelExecTimeThreshold;

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cout <<"\n Program failed after hipDeviceSynchronize()!\n\n";
		return 1; 
	}
	cout <<"Computation configuration: Threads per block="<<nThdsPerBlk<<", ["<<nBlks<<" BLOCKS, "<< BlkPerRow << " BLOCKS/ROW]\n";
	cout <<"\n--------------------------------------------------------------------------\n";
	cout <<GPUprop.name <<"    Compute Capability "<<GPUprop.major<<"."<<GPUprop.minor<<"  [max "<<SupportedBlocks<<" blocks; "<<maxThdsPerBlk<<" thr/blk] \n"; 
	cout <<"--------------------------------------------------------------------------\n"<<fixed<<setw(10)<<setprecision(2);
	cout <<"--------------------------------------------------------------------------\n";
	cout <<"            CPU->GPU Transfer   ="<<tfrCPUtoGPU<<" ms  ...  "<<set_size_MB(IMAGESIZE)<<" MB  ...  "<<setw(8)<<set_BW_GB(IMAGESIZE, tfrCPUtoGPU)<<" GB/s\n";
	cout <<"            GPU->CPU Transfer   ="<<tfrGPUtoCPU<<" ms  ...  "<<set_size_MB(IMAGESIZE)<<" MB  ...  "<<setw(8)<<set_BW_GB(IMAGESIZE, tfrGPUtoCPU)<<" GB/s\n";
	cout <<"----------------------------------------------------------------------------\n";
	cout <<"       BW Kernel Execution Time ="<<kernelExecTimeBW<<" ms  ...  "<<set_size_MB(GPUDataTfrBW)<<" MB  ...  "<< set_BW_GB(GPUDataTfrBW, kernelExecTimeBW)<<" GB/s\n";
	cout <<"    Gauss Kernel Execution Time ="<<kernelExecTimeGauss<<" ms  ...  "<<set_size_MB(GPUDataTfrGauss)<<" MB  ...  "<< set_BW_GB(GPUDataTfrGauss, kernelExecTimeGauss)<<" GB/s\n";
	cout <<"    Sobel Kernel Execution Time ="<<kernelExecTimeSobel<<" ms  ...  "<<set_size_MB(GPUDataTfrSobel)<<" MB  ...  "<< set_BW_GB(GPUDataTfrSobel, kernelExecTimeSobel)<<" GB/s\n";
	cout <<"Threshold Kernel Execution Time ="<<kernelExecTimeThreshold<<" ms  ...  "<<set_size_MB(GPUDataTfrThresh)<<" MB  ...  "<< set_BW_GB(GPUDataTfrThresh, kernelExecTimeThreshold)<<" GB/s\n";
	cout <<"----------------------------------------------------------------------------\n";
	cout <<"Total GPU kernel-only time      ="<<totalKernelTime<<" ms       "<< set_size_MB(GPUDataTfrKernel)<<" MB  ...  "<< setw(8)<<set_BW_GB(GPUDataTfrKernel, totalKernelTime)<<" GB/s\n";
	cout <<"Total time with I/O included    ="<<totalTime <<     " ms  ...  "<< set_size_MB(GPUDataTfrTotal) <<" MB  ...  "<<set_BW_GB(GPUDataTfrTotal, totalTime)<<" GB/s\n"; 
	cout <<"----------------------------------------------------------------------------\n";

	// Deallocate GPU memory and destroy events.
	hipFree(ptrGPU);
	hipEventDestroy(time1);	hipEventDestroy(time2);
	hipEventDestroy(time2BW);	hipEventDestroy(time2Gauss);	hipEventDestroy(time2Sobel);
	hipEventDestroy(time3);	hipEventDestroy(time4);
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools sunsigned char as Parallel Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
        cout <<"\ncudaDeviceReset failed!\n\n";
        return 1;
	}
    return 0;

}
